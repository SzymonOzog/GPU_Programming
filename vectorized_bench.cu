
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <cassert>
#include <hipblas.h>

#define BLOCK_SIZE 128 
#define BENCH_STEPS 1000
#define WARMUP_STEPS 100
#define VEC_RATIO 4
 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void clear_l2() {
    // Get actual L2 size via CUDA on first call of this function
    static int l2_clear_size = 0;
    static unsigned char* gpu_scratch_l2_clear = NULL;
    if (!gpu_scratch_l2_clear) {
        hipDeviceGetAttribute(&l2_clear_size, hipDeviceAttributeL2CacheSize, 0);
        l2_clear_size *= 2; // just to be extra safe (cache is not necessarily strict LRU)
        gpuErrchk(hipMalloc(&gpu_scratch_l2_clear, l2_clear_size));
    }
    // Clear L2 cache (this is run on every call unlike the above code)
    gpuErrchk(hipMemset(gpu_scratch_l2_clear, 0, l2_clear_size));
}

using datatype = float;
using datatype_vec = float4;

__global__ void copy(const int n , const datatype* __restrict__ in, datatype*  __restrict__ out)
{
  unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    out[i] = in[i];
  }
}

__global__ void copyh2(const int n , const datatype_vec* __restrict__ in, datatype_vec*  __restrict__ out)
{
  unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    out[i] = in[i];
  }
}

__global__ void copy_loop(const unsigned int n , const datatype* __restrict__ in, datatype*  __restrict__ out)
{
  unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
  for (int idx = i; idx < n; idx+=gridDim.x * blockDim.x)
  {
    out[idx] = in[idx];
  }
}

__global__ void copy_loop_float4(const unsigned int n , const datatype_vec* __restrict__ in, datatype_vec*  __restrict__ out)
{
  unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
  for (int idx = i; idx < n; idx+=gridDim.x * blockDim.x)
  {
    out[idx] = in[idx];
  }
}

int main()
{
  datatype* in_d;
  datatype* out_d;
  datatype* out2_d;

  long N = std::pow<long, long>(2, 25);

    //one warmup run
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    dim3 dimGrid(ceil(N/(float)BLOCK_SIZE), 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    hipMalloc((void**) &in_d, N*sizeof(datatype));
    datatype* cp = new datatype[N];
    for (int i = 0; i < N; i++)
    {
        cp[i] = (float)N;
    }
    hipMemcpy(in_d, cp, N*sizeof(datatype), hipMemcpyHostToDevice);

    hipMalloc((void**) &out_d, N*sizeof(datatype));
    hipMemset(out_d, 0, N*sizeof(datatype));

    hipMalloc((void**) &out2_d, N*sizeof(datatype));
    hipMemset(out2_d, 0, N*sizeof(datatype));
    float time = 0.f;
    double run_time = 0.0;
    for (int i = -WARMUP_STEPS; i<BENCH_STEPS; i++)
    {
      clear_l2();
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(start));
      copy<<<dimGrid, dimBlock>>>(N, in_d, out_d);
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time, start, stop));
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      if (i >= 0) // warmup
      {
        run_time += time / BENCH_STEPS;
      }
    }

    std::cout<<"regular time "<<run_time<<std::endl;

    dimGrid = dim3(ceil(N/(float)(BLOCK_SIZE*VEC_RATIO)), 1, 1);
    dimBlock = dim3(BLOCK_SIZE, 1, 1);
    time = 0.f;
    run_time = 0.0;
    for (int i = -WARMUP_STEPS; i<BENCH_STEPS; i++)
    {
      clear_l2();
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(start));
      copyh2<<<dimGrid, dimBlock>>>(N/VEC_RATIO, reinterpret_cast<datatype_vec*>(in_d), reinterpret_cast<datatype_vec*>(out_d));
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time, start, stop));
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      if (i >= 0) // warmup
      {
        run_time += time / BENCH_STEPS;
      }
    }

    std::cout<<"vectorized time "<<run_time<<std::endl;

    int loop_size = 1024;
    dimGrid = dim3(ceil(N/(float)(BLOCK_SIZE*loop_size)), 1, 1);
    dimBlock = dim3(BLOCK_SIZE, 1, 1);
    time = 0.f;
    run_time = 0.0;
    for (int i = -WARMUP_STEPS; i<BENCH_STEPS; i++)
    {
      clear_l2();
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(start));
      copy_loop<<<dimGrid, dimBlock>>>(N, in_d, out2_d);
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time, start, stop));
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      if (i >= 0) // warmup
      {
        run_time += time / BENCH_STEPS;
      }
    }

    datatype* out_h = new datatype[N];
    datatype* out2_h = new datatype[N];
    hipMemcpy(out_h, out_d, N*sizeof(datatype), hipMemcpyDeviceToHost);
    hipMemcpy(out2_h, out2_d, N*sizeof(datatype), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
      ASSERT(out_h[i] == out2_h[i], "failed at copy loop %d, %f, %f\n", i, (float)out_h[i], (float)out2_h[i]);
    }

    loop_size = loop_size/VEC_RATIO;
    std::cout<<"loop time "<<run_time<<std::endl;
    dimGrid = dim3(ceil(N/(float)(BLOCK_SIZE*VEC_RATIO*loop_size)), 1, 1);
    dimBlock = dim3(BLOCK_SIZE, 1, 1);
    time = 0.f;
    run_time = 0.0;
    for (int i = -WARMUP_STEPS; i<BENCH_STEPS; i++)
    {
      clear_l2();
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(start));
      copy_loop_float4<<<dimGrid, dimBlock>>>(N/VEC_RATIO, reinterpret_cast<datatype_vec*>(in_d), reinterpret_cast<datatype_vec*>(out2_d));
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time, start, stop));
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      if (i >= 0) // warmup
      {
        run_time += time / BENCH_STEPS;
      }
    }
    std::cout<<"loop time vectorized "<<run_time<<std::endl;

    hipMemcpy(out_h, out_d, N*sizeof(datatype), hipMemcpyDeviceToHost);
    hipMemcpy(out2_h, out2_d, N*sizeof(datatype), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
      ASSERT(out_h[i] == out2_h[i], "failed at  copy loopa datatype 4 %d, %f, %f\n", i, (float)out_h[i], (float)out2_h[i]);
    }

  hipFree(in_d);
  hipFree(out_d);
  return 0;
}

