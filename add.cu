
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int n , float* a, float* b, float* c)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    c[i] = a[i] + b[i];
  }
}

int main()
{
  int N = 4096;
  int BLOCK_SIZE=256;
  float* a = new float[N];
  float* b = new float[N];
  float* c = new float[N];
  for (int i = 0; i<N; i++)
  {
    a[i] = i;
    b[i] = 2*i;
  }
  float* a_d;
  float* b_d;
  float* c_d;

  hipMalloc((void**) &a_d, N*sizeof(float));
  hipMalloc((void**) &b_d, N*sizeof(float));
  hipMalloc((void**) &c_d, N*sizeof(float));

  hipMemcpy(a_d, a, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, N*sizeof(float), hipMemcpyHostToDevice);

  add<<<ceil(N/(float)BLOCK_SIZE), BLOCK_SIZE>>>(N, a_d, b_d, c_d);

  hipMemcpy(c, c_d, N*sizeof(float), hipMemcpyDeviceToHost);
  
  for (int i = 0; i<10; i++)
  {
    std::cout<<a[i]<<" "<<b[i]<<" "<<c[i]<<std::endl;
  }
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
  return 0;
}
