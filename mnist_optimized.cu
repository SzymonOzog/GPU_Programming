#include <chrono>
#include <cstring>
#include <fstream>
#include <iomanip>
#include <iostream> 
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <string>
#include <vector>

#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

class Timer
{
public:
  Timer(std::string in_name) : name(in_name)
  {
    start_time = std::chrono::system_clock::now();
  }
  ~Timer()
  {
    std::cout<<name<<" took "<<std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now() - start_time).count()/(float)1e6<<" ms"<<std::endl;
  }
private:
  std::chrono::time_point<std::chrono::system_clock> start_time;
  std::string name;
};

__global__ void forward(int batch_size, int n, int out_w, float* input, float* weights, float* biases, float* output)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    output[row*out_w+column] = biases[column];
    for(int i = 0; i < n; i++)
    {
      output[row*out_w+column] += weights[i*out_w + column] * input[row*n + i];
    }
  }
}

__global__ void forward_relu(int batch_size, int n, int out_w, float* input, float* weights, float* biases, float* output)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    float out = biases[column];
    for(int i = 0; i < n; i++)
    {
      out += weights[i*out_w + column] * input[row*n + i];
    }
    output[row*out_w+column] = out > 0.f ? out : 0.f;
  }
}

__global__ void backward(int batch_size, int n, int out_w, float* weights, float* biases, float* d_l, float* out_d_l, float* activations)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    float dl = 0.f;
    for(int i = 0; i < n; i++)
    {
      float w = weights[i*out_w + column];
      dl += w*d_l[row*n + i];
    }
    float activation = activations[row*out_w+column];
    out_d_l[row*out_w + column] = activation > 0.f ? dl : 0.f;
  }
}

__global__ void update_layer(int w, int h, int batch_size, float lr, float* weights, float* biases, float* activations, float* d_l)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    float dw = 0.f;
    float db = 0.f;
    for(int i = 0; i < batch_size; i++)
    {
      float act = activations[i*h + row];
      float dl = d_l[i*w + column];
      dw += act*dl;
      db += dl;
    }
    weights[row*w + column] -= lr * dw / batch_size;
    biases[column] -= lr * db / batch_size;
  }
}

__global__ void softmax(int w, int h, float* a, float* b)
{
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    float maxval = a[row*w];
    for (int i = 1; i<w; i++)
    {
      maxval = max(maxval, a[row*w + i]);
    }
    float divisor = 0.f;
    for (int i = 0; i<w; i++)
    {
      divisor += exp(a[row*w + i] - maxval);
    }
    b[row*w + col] = exp(a[row*w + col]-maxval)/(divisor);
  }
}

__global__ void cross_entropy(int w, int h, float* preds, float* real, float* output)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < h)
  {
    float loss = 0.f;
    for (int i = 0; i<w; i++)
    {
      loss -= real[idx*w + i] * log(max(1e-6, preds[idx*w + i]));
    }
    output[idx] = loss;
  }
}

__global__ void cross_entropy_backwards(int w, int h, float* preds, float* real, float* output)
{
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    output[row*w + col] = preds[row*w + col] - real[row*w + col];
  }
}

__global__ void init_rand(int w, int h, float* mat)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    hiprandState state;
    hiprand_init(44, row*w+column, 0, &state);
    mat[row*w + column] = hiprand_normal(&state)*sqrtf(2.f/h);
  }
}

void print_matrix(int w, int h, float* matrix, std::string title)
{
  float* m_h = new float[w*h];
  hipMemcpy(m_h, matrix, w*h*sizeof(float), hipMemcpyDeviceToHost);
  std::cout<<title<<std::endl;
  for(int i = 0; i<h; i++)
  {
    for(int j = 0; j<w; j++)
    {
      std::cout<<std::fixed<<std::setprecision(3)<<m_h[i*w+j]<<", ";
    }
    std::cout<<std::endl;
  }
  free(m_h);
}

void initLayer(float* weights, float* biases, int w, int h, int BLOCK_SIZE)
{
  dim3 dimGrid = dim3(ceil(w/(float)BLOCK_SIZE), ceil(h/(float)BLOCK_SIZE), 1);
  dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
  init_rand<<<dimGrid, dimBlock>>>(w, h, weights);
  gpuErrchk(hipPeekAtLastError());

  dimGrid = dim3(ceil(h/(float)BLOCK_SIZE), 1, 1);
  dimBlock = dim3(BLOCK_SIZE, 1, 1);
  init_rand<<<dimGrid, dimBlock>>>(1, h, biases);
  gpuErrchk(hipPeekAtLastError());
}

void read_mnist(std::ifstream& fin, int start, int length, float* x, float* y)
{
  constexpr int input_size = 784;
  constexpr int labels = 10;

  std::string line;
  std::vector<char> buffer(4096);

  for (int i = start; i < start + length; ++i)
  {
    if (!std::getline(fin, line)) {
      throw std::runtime_error("Unexpected end of file");
    }

    std::istringstream ss(line);

    int label;
    if (!(ss >> label)) {
      throw std::runtime_error("Failed to read label");
    }

    std::memset(y + labels * i, 0, labels * sizeof(float));
    y[labels * i + label] = 1.0f;

    float* x_row = x + i * input_size;
    for (int j = 0; j < input_size; ++j)
    {
      ASSERT(ss.getline(&buffer[0], buffer.size(), ','), "Failed to read pixel value for entry %d, pixel %d", i, j);
      x_row[j] = std::strtof(&buffer[0], nullptr) / 255.0f;
    }
  }
}

int main(int argc, char** argv)
{
  Timer full("full training");
  int test_length = 10000;
  int train_length = 60000;

  float* input;
  float* labels;
  int input_size = 784;
  int labels_size = 10;

  int BLOCK_SIZE = 16;
  int BATCH_SIZE = 64;
  int EPOCHS = 10;
  float LR = 0.03f;

  float* mnist_train_x = new float[input_size * train_length];
  float* mnist_train_y = new float[labels_size * train_length];

  float* mnist_test_x = new float[input_size * test_length];
  float* mnist_test_y = new float[labels_size * test_length];
  std::ifstream train_fin("./mnist_train.csv");
  std::ifstream test_fin("./mnist_test.csv");

  read_mnist(train_fin, 0, BATCH_SIZE, mnist_train_x, mnist_train_y);
  read_mnist(test_fin, 0, BATCH_SIZE, mnist_test_x, mnist_test_y);

  int size1 = 300;
  float* weights1;
  float* biases1;
  float* d_l1;

  int size2 = 100;
  float* weights2;
  float* biases2;
  float* d_l2;

  int size3 = 10;
  float* weights3;
  float* biases3;
  float* d_l3;


  dim3 dimGrid;
  dim3 dimBlock;

  float* out_h = new float[BATCH_SIZE*size3];
  float* loss_h = new float[BATCH_SIZE];

  float *x1;
  float *a1;
  float *x2;
  float *a2;
  float *x3;
  float *a3;
  float* loss;
  {
    Timer init("initialization");
    gpuErrchk(hipMalloc((void**) &input, input_size*BATCH_SIZE*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &labels, labels_size*BATCH_SIZE*sizeof(float)));

    gpuErrchk(hipMalloc((void**) &weights1, size1*input_size*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &biases1, size1*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &d_l1, size1*BATCH_SIZE*sizeof(float)));
    initLayer(weights1, biases1, size1, input_size, BLOCK_SIZE);

    gpuErrchk(hipMalloc((void**) &weights2, size2*size1*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &biases2, size2*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &d_l2, size2*BATCH_SIZE*sizeof(float)));
    initLayer(weights2, biases2, size2, size1, BLOCK_SIZE);


    gpuErrchk(hipMalloc((void**) &weights3, size3*size2*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &biases3, size3*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &d_l3, size3*BATCH_SIZE*sizeof(float)));
    initLayer(weights3, biases3, size3, size2, BLOCK_SIZE);

    gpuErrchk(hipMalloc((void**) &x1, size1*BATCH_SIZE*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &a1, size1*BATCH_SIZE*sizeof(float)));

    gpuErrchk(hipMalloc((void**) &x2, size2*BATCH_SIZE*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &a2, size2*BATCH_SIZE*sizeof(float)));

    gpuErrchk(hipMalloc((void**) &x3, size3*BATCH_SIZE*sizeof(float)));
    gpuErrchk(hipMalloc((void**) &a3, size3*BATCH_SIZE*sizeof(float)));

    gpuErrchk(hipMalloc((void**) &loss, BATCH_SIZE*sizeof(float)));
  }

  float total_time = 0.f;
  for(int epoch = 0; epoch<EPOCHS; epoch++)
  {
    float cum_loss = 0.f;
    int correct = 0;
    int total = 0;
    auto start_time = std::chrono::system_clock::now();
    for(int batch = 0; batch<train_length/BATCH_SIZE; batch++)
    {
      total += BATCH_SIZE;
      gpuErrchk(hipMemcpy(input, &mnist_train_x[batch*BATCH_SIZE*input_size], BATCH_SIZE*input_size*sizeof(float), hipMemcpyHostToDevice)); 
      gpuErrchk(hipMemcpy(labels, &mnist_train_y[batch*BATCH_SIZE*labels_size], BATCH_SIZE*labels_size*sizeof(float), hipMemcpyHostToDevice)); 

      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward_relu<<<dimGrid, dimBlock>>>(BATCH_SIZE, input_size, size1, input, weights1, biases1, a1);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward_relu<<<dimGrid, dimBlock>>>(BATCH_SIZE, size1, size2, a1, weights2, biases2, a2);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size2, size3, a2, weights3, biases3, x3);
      gpuErrchk(hipPeekAtLastError());

      softmax<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, x3, a3);
      gpuErrchk(hipPeekAtLastError());
      
      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), 1, 1);
      dimBlock = dim3(BLOCK_SIZE, 1, 1);
      cross_entropy<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, a3, labels, loss);

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      cross_entropy_backwards<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, a3, labels, d_l3);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      backward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size3, size2, weights3, biases3, d_l3, d_l2, a2);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      backward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size2, size1, weights2, biases2, d_l2, d_l1, a1);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(size2/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
      update_layer<<<dimGrid, dimBlock>>>(size3, size2, BATCH_SIZE, LR, weights3, biases3, a2, d_l3);
      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(size1/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
      update_layer<<<dimGrid, dimBlock>>>(size2, size1, BATCH_SIZE, LR, weights2, biases2, a1, d_l2);
      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(input_size/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
      update_layer<<<dimGrid, dimBlock>>>(size1, input_size, BATCH_SIZE, LR, weights1, biases1, input, d_l1);

      if (epoch == 0 && (batch+2)*BATCH_SIZE < train_length)
      {
        read_mnist(train_fin, (batch+1)*BATCH_SIZE, BATCH_SIZE, mnist_train_x, mnist_train_y);
      }

      gpuErrchk(hipMemcpy(out_h, a3, BATCH_SIZE*size3*sizeof(float), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(loss_h, loss, BATCH_SIZE*sizeof(float), hipMemcpyDeviceToHost));
      
      for (int i = 0; i < BATCH_SIZE; i++)
      {
        float max_1 = 0.f;
        float max_2 = 0.f;
        int i1 = 0;
        int i2 = 0;
        for (int j = 0; j<labels_size; j++)
        {
          if (out_h[i*labels_size + j] > max_1)
          {
            max_1 = out_h[i*labels_size + j];
            i1 = j;
          }
          
          if (mnist_train_y[batch*BATCH_SIZE*labels_size + i*labels_size + j] > max_2)
          {
            max_2 = mnist_train_y[batch*BATCH_SIZE*labels_size + i*labels_size + j];
            i2 = j;
          }
        }
        correct += (i1 == i2);
        cum_loss += loss_h[i];
      }

    }
    float val_loss = 0.f;
    int val_correct = 0;
    int val_total = 0;
    for(int batch = 0; batch<test_length/BATCH_SIZE; batch++)
    {
      val_total += BATCH_SIZE;
      gpuErrchk(hipMemcpy(input, &mnist_test_x[batch*BATCH_SIZE*input_size], BATCH_SIZE*input_size*sizeof(float), hipMemcpyHostToDevice)); 
      gpuErrchk(hipMemcpy(labels, &mnist_test_y[batch*BATCH_SIZE*labels_size], BATCH_SIZE*labels_size*sizeof(float), hipMemcpyHostToDevice)); 

      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward_relu<<<dimGrid, dimBlock>>>(BATCH_SIZE, input_size, size1, input, weights1, biases1, a1);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward_relu<<<dimGrid, dimBlock>>>(BATCH_SIZE, size1, size2, a1, weights2, biases2, a2);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size2, size3, a2, weights3, biases3, x3);
      gpuErrchk(hipPeekAtLastError());

      softmax<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, x3, a3);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), 1, 1);
      dimBlock = dim3(BLOCK_SIZE, 1, 1);
      cross_entropy<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, a3, labels, loss);

      if (epoch == 0 && (batch+2)*BATCH_SIZE < test_length)
      {
        read_mnist(test_fin, (batch+1)*BATCH_SIZE, BATCH_SIZE, mnist_test_x, mnist_test_y);
      }
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipMemcpy(out_h, a3, BATCH_SIZE*size3*sizeof(float), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(loss_h, loss, BATCH_SIZE*sizeof(float), hipMemcpyDeviceToHost));
      
      for (int i = 0; i < BATCH_SIZE; i++)
      {
        float max_1 = 0.f;
        float max_2 = 0.f;
        int i1 = 0;
        int i2 = 0;
        for (int j = 0; j<labels_size; j++)
        {
          if (out_h[i*labels_size + j] > max_1)
          {
            max_1 = out_h[i*labels_size + j];
            i1 = j;
          }
          
          if (mnist_test_y[batch*BATCH_SIZE*labels_size + i*labels_size + j] > max_2)
          {
            max_2 = mnist_test_y[batch*BATCH_SIZE*labels_size + i*labels_size + j];
            i2 = j;
          }
        }
        val_correct += (i1 == i2);
        val_loss += loss_h[i];
      }
    }

    float epoch_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now() - start_time).count();
    total_time += epoch_time;
    std::cout<<"epoch "<<epoch<<" took "<<epoch_time<<
      "ms cum loss "<<cum_loss<<" accuracy "<<(float)correct/total<<
      " val loss "<<val_loss<<" val accuracy "<<(float)val_correct/val_total<<std::endl;
  }
  std::cout<<"finished training, total time = "<<total_time<<" ms"<<std::endl;
}
