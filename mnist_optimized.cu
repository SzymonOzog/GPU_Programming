#include <chrono>
#include <fstream>
#include <iomanip>
#include <iostream> 
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <string>

#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void forward(int batch_size, int n, int out_w, float* input, float* weights, float* biases, float* output)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    output[row*out_w+column] = biases[column];
    for(int i = 0; i < n; i++)
    {
      output[row*out_w+column] += weights[i*out_w + column] * input[row*n + i];
    }
  }
}

__global__ void backward(int batch_size, int n, int out_w, float* weights, float* biases, float* d_l, float* out_d_l, float* activations)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    float dl = 0.f;
    for(int i = 0; i < n; i++)
    {
      float w = weights[i*out_w + column];
      dl += w*d_l[row*n + i];
    }
    float activation = activations[row*out_w+column];
    out_d_l[row*out_w + column] = activation > 0.f ? dl : 0.f;
  }
}

__global__ void update_layer(int w, int h, int batch_size, float lr, float* weights, float* biases, float* activations, float* d_l)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    float dw = 0.f;
    float db = 0.f;
    for(int i = 0; i < batch_size; i++)
    {
      float act = activations[i*h + row];
      float dl = d_l[i*w + column];
      dw += act*dl;
      db += dl;
    }
    weights[row*w + column] -= lr * dw / batch_size;
    biases[column] -= lr * db / batch_size;
  }
}

__global__ void relu(int w, int h, float* a, float* b)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < w && column < h)
  {
    float activation = a[row*w+column];
    b[row*w+column] =  activation > 0.f ? activation : 0.f;
  }
}

__global__ void relu_backwards(int w, int h, int ns, float* a, float* d_l, float* b)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < w && column < h)
  {
    float activation = a[row*w+column];
    b[row*w+column] = activation > 0.f ? d_l[row*w+column] : 0.f;
  }
}

__global__ void softmax(int w, int h, float* a, float* b)
{
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    float maxval = a[row*w];
    for (int i = 1; i<w; i++)
    {
      maxval = max(maxval, a[row*w + i]);
    }
    float divisor = 0.f;
    for (int i = 0; i<w; i++)
    {
      divisor += exp(a[row*w + i] - maxval);
    }
    b[row*w + col] = exp(a[row*w + col]-maxval)/(divisor);
  }
}

__global__ void cross_entropy(int w, int h, float* preds, float* real, float* output)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < h)
  {
    float loss = 0.f;
    for (int i = 0; i<w; i++)
    {
      loss -= real[idx*w + i] * log(max(1e-6, preds[idx*w + i]));
    }
    output[idx] = loss;
  }
}

__global__ void cross_entropy_backwards(int w, int h, float* preds, float* real, float* output)
{
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    output[row*w + col] = preds[row*w + col] - real[row*w + col];
  }
}

__global__ void init_rand(int w, int h, float* mat)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    hiprandState state;
    hiprand_init(44, row*w+column, 0, &state);
    mat[row*w + column] = hiprand_normal(&state)*sqrtf(2.f/h);
  }
}

void print_matrix(int w, int h, float* matrix, std::string title)
{
  float* m_h = new float[w*h];
  hipMemcpy(m_h, matrix, w*h*sizeof(float), hipMemcpyDeviceToHost);
  std::cout<<title<<std::endl;
  for(int i = 0; i<h; i++)
  {
    for(int j = 0; j<w; j++)
    {
      std::cout<<std::fixed<<std::setprecision(3)<<m_h[i*w+j]<<", ";
    }
    std::cout<<std::endl;
  }
  free(m_h);
}

void initLayer(float* weights, float* biases, int w, int h, int BLOCK_SIZE)
{
  dim3 dimGrid = dim3(ceil(w/(float)BLOCK_SIZE), ceil(h/(float)BLOCK_SIZE), 1);
  dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
  init_rand<<<dimGrid, dimBlock>>>(w, h, weights);
  gpuErrchk(hipPeekAtLastError());

  dimGrid = dim3(ceil(h/(float)BLOCK_SIZE), 1, 1);
  dimBlock = dim3(BLOCK_SIZE, 1, 1);
  init_rand<<<dimGrid, dimBlock>>>(1, h, biases);
  gpuErrchk(hipPeekAtLastError());
}

void read_mnist(const std::string filename, int length, float* x, float* y)
{
  int input_size = 784;
  int labels = 10;

  std::fstream fin;
  fin.open("./mnist_train.csv");
  std::string row;
  constexpr char delim = ',';
  for(int i = 0; i<length; i++)
  {
    fin >> row;
    int pos = row.find(delim);
    int label = std::stoi(row.substr(0, pos+1));
    for(int j = 0; j<labels; j++)
    {
      y[labels*i + j] = (j==label);
    }
    row.erase(0, pos+1);
    for(int j = 0; j<input_size; j++)
    {
      pos = row.find(delim);
      if (pos == std::string::npos)
      {
        pos = row.length() - 1;
      }
      x[i*input_size+j] = std::stof(row.substr(0, pos+1)) / 255; //normalize value
      row.erase(0, pos+1);
    }
    ASSERT(row.length() == 0, "didn't parse all values in row, %d", i);
  }
}

int main(int argc, char** argv)
{
  int test_length = 10000;
  int train_length = 60000;

  float* input;
  float* labels;
  int input_size = 784;
  int labels_size = 10;

  float* mnist_train_x = new float[input_size * train_length];
  float* mnist_train_y = new float[labels_size * train_length];
  read_mnist("./mnist_train.csv", train_length, mnist_train_x, mnist_train_y);

  float* mnist_test_x = new float[input_size * test_length];
  float* mnist_test_y = new float[labels_size * test_length];
  read_mnist("./mnist_test.csv", test_length, mnist_test_x, mnist_test_y);

  int size1 = 300;
  float* weights1;
  float* biases1;
  float* d_l1;

  int size2 = 100;
  float* weights2;
  float* biases2;
  float* d_l2;

  int size3 = 10;
  float* weights3;
  float* biases3;
  float* d_l3;


  int BLOCK_SIZE = 16;
  int BATCH_SIZE = 16;
  int EPOCHS = 3000;
  float LR = 0.003f;
  dim3 dimGrid;
  dim3 dimBlock;

  float* out_h = new float[BATCH_SIZE*size3];
  float* loss_h = new float[BATCH_SIZE];


  gpuErrchk(hipMalloc((void**) &input, input_size*BATCH_SIZE*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &labels, labels_size*BATCH_SIZE*sizeof(float)));

  gpuErrchk(hipMalloc((void**) &weights1, size1*input_size*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &biases1, size1*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &d_l1, size1*BATCH_SIZE*sizeof(float)));
  initLayer(weights1, biases1, size1, input_size, BLOCK_SIZE);

  gpuErrchk(hipMalloc((void**) &weights2, size2*size1*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &biases2, size2*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &d_l2, size2*BATCH_SIZE*sizeof(float)));
  initLayer(weights2, biases2, size2, size1, BLOCK_SIZE);


  gpuErrchk(hipMalloc((void**) &weights3, size3*size2*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &biases3, size3*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &d_l3, size3*BATCH_SIZE*sizeof(float)));
  initLayer(weights3, biases3, size3, size2, BLOCK_SIZE);

  float *x1;
  float *a1;
  gpuErrchk(hipMalloc((void**) &x1, size1*BATCH_SIZE*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &a1, size1*BATCH_SIZE*sizeof(float)));

  float *x2;
  float *a2;
  gpuErrchk(hipMalloc((void**) &x2, size2*BATCH_SIZE*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &a2, size2*BATCH_SIZE*sizeof(float)));

  float *x3;
  float *a3;
  gpuErrchk(hipMalloc((void**) &x3, size3*BATCH_SIZE*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &a3, size3*BATCH_SIZE*sizeof(float)));
      
  float* loss;
  gpuErrchk(hipMalloc((void**) &loss, BATCH_SIZE*sizeof(float)));


  for(int epoch = 0; epoch<EPOCHS; epoch++)
  {
    float cum_loss = 0.f;
    int correct = 0;
    int total = 0;
    auto start_time = std::chrono::system_clock::now();
    for(int batch = 0; batch<train_length/BATCH_SIZE; batch++)
    {
      total += BATCH_SIZE;
      gpuErrchk(hipMemcpy(input, &mnist_train_x[batch*BATCH_SIZE*input_size], BATCH_SIZE*input_size*sizeof(float), hipMemcpyHostToDevice)); 
      gpuErrchk(hipMemcpy(labels, &mnist_train_y[batch*BATCH_SIZE*labels_size], BATCH_SIZE*labels_size*sizeof(float), hipMemcpyHostToDevice)); 

      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, input_size, size1, input, weights1, biases1, x1);
      gpuErrchk(hipPeekAtLastError());

      relu<<<dimGrid, dimBlock>>>(size1, BATCH_SIZE, x1, a1);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size1, size2, a1, weights2, biases2, x2);
      gpuErrchk(hipPeekAtLastError());

      relu<<<dimGrid, dimBlock>>>(size2, BATCH_SIZE, x2, a2);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size2, size3, a2, weights3, biases3, x3);
      gpuErrchk(hipPeekAtLastError());

      softmax<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, x3, a3);
      gpuErrchk(hipPeekAtLastError());
      
      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), 1, 1);
      dimBlock = dim3(BLOCK_SIZE, 1, 1);
      cross_entropy<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, a3, labels, loss);

      gpuErrchk(hipDeviceSynchronize());

      gpuErrchk(hipMemcpy(out_h, a3, BATCH_SIZE*size3*sizeof(float), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(loss_h, loss, BATCH_SIZE*sizeof(float), hipMemcpyDeviceToHost));
      
      for (int i = 0; i < BATCH_SIZE; i++)
      {
        float max_1 = 0.f;
        float max_2 = 0.f;
        int i1 = 0;
        int i2 = 0;
        for (int j = 0; j<labels_size; j++)
        {
          if (out_h[i*labels_size + j] > max_1)
          {
            max_1 = out_h[i*labels_size + j];
            i1 = j;
          }
          
          if (mnist_train_y[batch*BATCH_SIZE*labels_size + i*labels_size + j] > max_2)
          {
            max_2 = mnist_train_y[batch*BATCH_SIZE*labels_size + i*labels_size + j];
            i2 = j;
          }
        }
        correct += (i1 == i2);
        cum_loss += loss_h[i];
      }

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      cross_entropy_backwards<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, a3, labels, d_l3);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      backward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size3, size2, weights3, biases3, d_l3, d_l2, a2);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      backward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size2, size1, weights2, biases2, d_l2, d_l1, a1);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(size2/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
      update_layer<<<dimGrid, dimBlock>>>(size3, size2, BATCH_SIZE, LR, weights3, biases3, a2, d_l3);
      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(size1/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
      update_layer<<<dimGrid, dimBlock>>>(size2, size1, BATCH_SIZE, LR, weights2, biases2, a1, d_l2);
      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(input_size/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
      update_layer<<<dimGrid, dimBlock>>>(size1, input_size, BATCH_SIZE, LR, weights1, biases1, input, d_l1);

    }
    float val_loss = 0.f;
    int val_correct = 0;
    int val_total = 0;
    for(int batch = 0; batch<test_length/BATCH_SIZE; batch++)
    {
      val_total += BATCH_SIZE;
      gpuErrchk(hipMemcpy(input, &mnist_test_x[batch*BATCH_SIZE*input_size], BATCH_SIZE*input_size*sizeof(float), hipMemcpyHostToDevice)); 
      gpuErrchk(hipMemcpy(labels, &mnist_test_y[batch*BATCH_SIZE*labels_size], BATCH_SIZE*labels_size*sizeof(float), hipMemcpyHostToDevice)); 

      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, input_size, size1, input, weights1, biases1, x1);
      gpuErrchk(hipPeekAtLastError());

      relu<<<dimGrid, dimBlock>>>(size1, BATCH_SIZE, x1, a1);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size1, size2, a1, weights2, biases2, x2);
      gpuErrchk(hipPeekAtLastError());

      relu<<<dimGrid, dimBlock>>>(size2, BATCH_SIZE, x2, a2);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size2, size3, a2, weights3, biases3, x3);
      gpuErrchk(hipPeekAtLastError());

      softmax<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, x3, a3);
      gpuErrchk(hipPeekAtLastError());

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), 1, 1);
      dimBlock = dim3(BLOCK_SIZE, 1, 1);
      cross_entropy<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, a3, labels, loss);

      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipMemcpy(out_h, a3, BATCH_SIZE*size3*sizeof(float), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(loss_h, loss, BATCH_SIZE*sizeof(float), hipMemcpyDeviceToHost));
      
      for (int i = 0; i < BATCH_SIZE; i++)
      {
        float max_1 = 0.f;
        float max_2 = 0.f;
        int i1 = 0;
        int i2 = 0;
        for (int j = 0; j<labels_size; j++)
        {
          if (out_h[i*labels_size + j] > max_1)
          {
            max_1 = out_h[i*labels_size + j];
            i1 = j;
          }
          
          if (mnist_test_y[batch*BATCH_SIZE*labels_size + i*labels_size + j] > max_2)
          {
            max_2 = mnist_test_y[batch*BATCH_SIZE*labels_size + i*labels_size + j];
            i2 = j;
          }
        }
        val_correct += (i1 == i2);
        val_loss += loss_h[i];
      }
    }

    auto time_total = std::chrono::system_clock::now() - start_time;
    std::cout<<"epoch "<<epoch<<" took "<<std::chrono::duration_cast<std::chrono::milliseconds>(time_total).count()<<
      "ms cum loss "<<cum_loss<<" accuracy "<<(float)correct/total<<
      " val loss "<<val_loss<<" val accuracy "<<(float)val_correct/val_total<<std::endl;
  }
}
