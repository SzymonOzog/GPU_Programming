
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <chrono>
#include <random>

#define TILE_WIDTH 32
#define BENCH_STEPS 3
#define TIMINGS 8
#define START 8

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void matmul_elem(int n, float* a, float* b, float* c)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < n && column < n)
  {
    float dot_prod = 0.f;
    for(int i = 0; i < n; i++)
    {
      dot_prod += a[row*n + i] * b[i*n + column];
    }
    c[row*n+column] = dot_prod;
  }
}

__global__ void tiled_matmul(int n, float* a, float* b, float* c)
{
  __shared__ float a_tile[TILE_WIDTH][TILE_WIDTH];
  __shared__ float b_tile[TILE_WIDTH][TILE_WIDTH];

  int column = blockIdx.x*TILE_WIDTH + threadIdx.x;
  int row = blockIdx.y*TILE_WIDTH + threadIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  float dot_prod = 0.f;
  for (int tile_offset = 0; tile_offset<n; tile_offset+=TILE_WIDTH)
  {
    int a_chk = tile_offset+tx < n && row < n;
    a_tile[ty][tx] = a_chk ? a[row*n + tile_offset+tx] : 0.f;

    int b_chk = (tile_offset+ty) < n && column < n;
    b_tile[ty][tx] = b_chk ? b[(tile_offset+ty)*n + column] : 0.f;

    __syncthreads();
    for(int i = 0; i < TILE_WIDTH; i++)
    {
      dot_prod += a_tile[ty][i] * b_tile[i][tx];
    }
    __syncthreads();
  }

  if (row < n && column < n)
  {
    c[row*n+column] = dot_prod;
  }
}

float get_random()
{
    static std::default_random_engine e;
    static std::uniform_real_distribution<> dis(0, 1); // range [0, 1)
    return dis(e);
}

void cpu_matmul(int n, float* a, float* b, float*c)
{
  for (int i = 0; i<n; i++)
  {
    for (int j = 0; j<n; j++)
    {
      float dot_product = 0.f;
      for (int k = 0; k<n; k++)
      {
        dot_product += a[i*n + k] * b[k*n + j];
      }
      c[i*n+j] = dot_product; 
    }
  }
}

int main()
{
  float mt[TIMINGS];
  float tt[TIMINGS];
  float ct[TIMINGS];
  float* a_d;
  float* b_d;
  float* c_d;
  float* d_d;

  long max_N = std::pow<long, long>(2, START+TIMINGS-1);
  hipMalloc((void**) &a_d, max_N*max_N*sizeof(float));
  hipMalloc((void**) &b_d, max_N*max_N*sizeof(float));
  hipMalloc((void**) &c_d, max_N*max_N*sizeof(float));
  hipMalloc((void**) &d_d, max_N*max_N*sizeof(float));

  float* a = new float[max_N * max_N];
  float* b = new float[max_N * max_N];
  float* c = new float[max_N * max_N];

  for (int p = START; p<START+TIMINGS; p++)
  {
    long N = std::pow<long, long>(2, p);
    int BLOCK_SIZE=32;

    dim3 dimGrid(ceil(N/(float)BLOCK_SIZE), ceil(N/(float)BLOCK_SIZE), 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    double matmul_time=0.0;
    for (int i = -1; i<BENCH_STEPS; i++)
    {
      // CLEAR CACHE
      hipMemset(a_d, 1, max_N*max_N*sizeof(float));
      hipMemset(b_d, 1, max_N*max_N*sizeof(float));
      hipMemset(c_d, 1, max_N*max_N*sizeof(float));
      hipMemset(d_d, 1, max_N*max_N*sizeof(float));
      auto start_time = std::chrono::system_clock::now();
      matmul_elem<<<dimGrid, dimBlock>>>(N, a_d, b_d, c_d);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      double final_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now() - start_time).count();
      if (i != -1) // one warmup run
      {
        matmul_time += final_time;
      }
    }

    dimGrid = dim3(ceil(N/(float)TILE_WIDTH), ceil(N/(float)TILE_WIDTH), 1);
    dimBlock = dim3(TILE_WIDTH, TILE_WIDTH, 1);

    double tiled_time=0.0;
    for (int i = -1; i<BENCH_STEPS; i++)
    {
      // CLEAR CACHE
      hipMemset(a_d, 1, max_N*max_N*sizeof(float));
      hipMemset(b_d, 1, max_N*max_N*sizeof(float));
      hipMemset(c_d, 1, max_N*max_N*sizeof(float));
      hipMemset(d_d, 1, max_N*max_N*sizeof(float));
      auto start_time = std::chrono::system_clock::now();
      tiled_matmul<<<dimGrid, dimBlock>>>(N, a_d, b_d, d_d);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      double final_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now() - start_time).count();
      if (i != -1) // one warmup run
      {
        tiled_time += final_time;
      }
    }

    double cpu_time=0.0;
    for (int i = -1; i<BENCH_STEPS; i++)
    {
      // CLEAR CACHE
      memset(a, 1, max_N*max_N*sizeof(float));
      memset(b, 1, max_N*max_N*sizeof(float));
      memset(c, 1, max_N*max_N*sizeof(float));
      auto start_time = std::chrono::system_clock::now();
      cpu_matmul(N, a, b, c);
      double final_time = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now() - start_time).count();
      if (i != -1) // one warmup run
      {
        cpu_time += final_time;
      }
    }
    std::cout<<"n = "<<N<<" matmul time: "<<matmul_time/BENCH_STEPS<<" tiled time: "<<tiled_time/BENCH_STEPS<<" cpu time: "<<cpu_time/BENCH_STEPS<<std::endl;

    mt[p-START] = matmul_time/BENCH_STEPS;
    tt[p-START] = tiled_time/BENCH_STEPS;
    ct[p-START] = cpu_time/BENCH_STEPS;
  }
  float* c_h = new float[max_N*max_N];
  float* d_h = new float[max_N*max_N];
  hipMemcpy(c_h, c_d, max_N*max_N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(d_h, d_d, max_N*max_N*sizeof(float), hipMemcpyDeviceToHost);
  float tolerance = 1e-6;
  for (int i = 0; i < max_N*max_N; i++)
  {
    ASSERT(abs(c[i] - d_h[i]) < tolerance, "failed at %d, %f, %f\n", i, c[i], d_h[i]);
  }
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
  hipFree(d_d);

  std::cout<<"normal_times = [";
  for (int i = 0; i<TIMINGS; i++)
  {
    std::cout<<mt[i]<<", ";
  }
  std::cout<<"]"<<std::endl;

  std::cout<<"tiled_times = [";
  for (int i = 0; i<TIMINGS; i++)
  {
    std::cout<<tt[i]<<", ";
  }
  std::cout<<"]"<<std::endl;

  std::cout<<"cpu_times = [";
  for (int i = 0; i<TIMINGS; i++)
  {
    std::cout<<ct[i]<<", ";
  }
  std::cout<<"]"<<std::endl;
  return 0;
}
