
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <cassert>

#define BLOCK_SIZE 1024
#define CONST_SIZE 16384
#define BENCH_STEPS 1000
#define TIMINGS 14
#define START 10
#define ACCESSES 10

#define access (threadIdx.x * dist) % CONST_SIZE

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void clear_l2() {
    // Get actual L2 size via CUDA on first call of this function
    static int l2_clear_size = 0;
    static unsigned char* gpu_scratch_l2_clear = NULL;
    if (!gpu_scratch_l2_clear) {
        hipDeviceGetAttribute(&l2_clear_size, hipDeviceAttributeL2CacheSize, 0);
        l2_clear_size *= 2; // just to be extra safe (cache is not necessarily strict LRU)
        gpuErrchk(hipMalloc(&gpu_scratch_l2_clear, l2_clear_size));
    }
    // Clear L2 cache (this is run on every call unlike the above code)
    gpuErrchk(hipMemset(gpu_scratch_l2_clear, 0, l2_clear_size));
}

__constant__ float c_mem[CONST_SIZE];

__global__ void add(int n , float* a, float* b, float* c, int dist)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int y = access;
  if (i < n-ACCESSES)
  {
    for(int x = 0; x<ACCESSES; x++)
    {
      c[i] = a[i] + b[y+x];
    }
  }
}

__global__ void add_const(int n , float* a, float* c, int dist)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int y = access;
  if (i < n-ACCESSES)
  {
    for(int x = 0; x<ACCESSES; x++)
    {
      c[i] = a[i] + c_mem[y+x];
    }
  }
}

int main()
{
  float mt[TIMINGS];
  float tt[TIMINGS];
  float* a_d;
  float* b_d;
  float* c_d;
  float* d_d;

  long max_N = std::pow<long, long>(2, START+TIMINGS-1);
  hipMalloc((void**) &a_d, max_N*sizeof(float));
  hipMalloc((void**) &b_d, CONST_SIZE*sizeof(float));
  hipMalloc((void**) &c_d, max_N*sizeof(float));
  hipMalloc((void**) &d_d, max_N*sizeof(float));

  float* cmemset = new float[max_N];
  hipMemset(a_d, 1, max_N*sizeof(float));
  hipMemset(b_d, 1, CONST_SIZE*sizeof(float));
  memset(cmemset, 1, CONST_SIZE*sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(c_mem), cmemset, CONST_SIZE*sizeof(float));
  hipMemset(d_d, 1, max_N*sizeof(float));

  for (int distance = 1; distance<17; distance++)
  {
    for (int p = START; p<START+TIMINGS; p++)
    {
      hipEvent_t start, stop;
      float time;
      gpuErrchk(hipEventCreate(&start));
      gpuErrchk(hipEventCreate(&stop));
      long N = std::pow<long, long>(2, p);

      dim3 dimGrid(ceil(N/(float)BLOCK_SIZE), 1, 1);
      dim3 dimBlock(BLOCK_SIZE, 1, 1);

      double add_time=0.0;
      for (int i = -1; i<BENCH_STEPS; i++)
      {
        clear_l2();
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipEventRecord(start));
        add<<<dimGrid, dimBlock>>>(N, a_d, b_d, c_d, distance);
        gpuErrchk(hipEventRecord(stop));
        gpuErrchk(hipEventSynchronize(stop));
        gpuErrchk(hipEventElapsedTime(&time, start, stop));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        if (i != -1) // one warmup run
        {
          add_time += time / BENCH_STEPS;
        }
      }

      double const_time=0.0;
      for (int i = -1; i<BENCH_STEPS; i++)
      {
        clear_l2();
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipEventRecord(start));
        add_const<<<dimGrid, dimBlock>>>(N, a_d, d_d, distance);
        gpuErrchk(hipEventRecord(stop));
        gpuErrchk(hipEventSynchronize(stop));
        gpuErrchk(hipEventElapsedTime(&time, start, stop));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        if (i != -1) // one warmup run
        {
          const_time += time / BENCH_STEPS;
        }
      }

      mt[p-START] = add_time;
      tt[p-START] = const_time;
      gpuErrchk(hipEventDestroy(start));
      gpuErrchk(hipEventDestroy(stop));
    }
    float* c_h = new float[max_N];
    float* d_h = new float[max_N];
    hipMemcpy(c_h, c_d, max_N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_h, d_d, max_N*sizeof(float), hipMemcpyDeviceToHost);
    float tolerance = 1e-6;
    for (int i = 0; i < max_N; i++)
    {
      ASSERT(abs(c_h[i] - d_h[i]) < tolerance, "failed at %d, %f, %f\n", i, c_h[i], d_h[i]);
    }
    std::cout<<"ratio"<<distance<<" = [";
    for (int i = 0; i<TIMINGS; i++)
    {
      std::cout<<std::fixed<<std::setprecision(3)<<tt[i]/mt[i]<<", ";
    }
    std::cout<<"]"<<std::endl;
  }
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
  hipFree(d_d);
  return 0;
}

