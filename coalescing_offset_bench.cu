
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <cassert>

#define BLOCK_SIZE 32 
#define BENCH_STEPS 4000
#define MAX_OFFSET 129 
 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void clear_l2() {
    // Get actual L2 size via CUDA on first call of this function
    static int l2_clear_size = 0;
    static unsigned char* gpu_scratch_l2_clear = NULL;
    if (!gpu_scratch_l2_clear) {
        hipDeviceGetAttribute(&l2_clear_size, hipDeviceAttributeL2CacheSize, 0);
        l2_clear_size *= 2; // just to be extra safe (cache is not necessarily strict LRU)
        gpuErrchk(hipMalloc(&gpu_scratch_l2_clear, l2_clear_size));
    }
    // Clear L2 cache (this is run on every call unlike the above code)
    gpuErrchk(hipMemset(gpu_scratch_l2_clear, 0, l2_clear_size));
}

__global__ void copy(int n , float* in, float* out, int offset)
{
  unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    out[i + offset] = in[i + offset];
  }
}

int main()
{
  double timings[MAX_OFFSET];
  float* in_d;
  float* out_d;

  long N = std::pow<long, long>(2, 20);

  for (int o = -1; o<MAX_OFFSET; o++)
  {
    //one warmup run
    int offset = std::max(o, 0);
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    dim3 dimGrid(ceil(N/(float)BLOCK_SIZE), 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    hipMalloc((void**) &in_d, (N+offset)*sizeof(float));
    hipMalloc((void**) &out_d, (N+offset)*sizeof(float));
    float time = 0.f;
    double run_time = 0.0;
    for (int i = -1; i<BENCH_STEPS; i++)
    {
      clear_l2();
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(start));
      copy<<<dimGrid, dimBlock>>>(N, in_d, out_d, offset);
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time, start, stop));
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      if (i != -1) // one warmup run
      {
        run_time += time / BENCH_STEPS;
      }
    }

    timings[offset] = run_time;
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));
  }
  std::cout<<"timings"<<" = [";
  for (int i = 0; i<MAX_OFFSET; i++)
  {
    std::cout<<std::fixed<<std::setprecision(6)<<timings[i]<<", ";
  }
  std::cout<<"]"<<std::endl;
  hipFree(in_d);
  hipFree(out_d);
  return 0;
}

