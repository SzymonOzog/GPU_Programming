#include <fstream>
#include <iomanip>
#include <iostream> 
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <random>
#include <string>

#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void forward(int batch_size, int n, int out_w, float* input, float* weights, float* biases, float* output)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    output[row*out_w+column] = biases[column];
    for(int i = 0; i < n; i++)
    {
      output[row*out_w+column] += weights[i*out_w + column] * input[row*n + i];
    }
  }
}

__global__ void backward(int batch_size, int n, int out_w, float lr, float* weights, float* biases, float* d_l, float* out_d_l)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    float dl = 0.f;
    for(int i = 0; i < n; i++)
    {
      float w = weights[i*out_w + column];
      float dl = d_l[row*n + i];
      dl += w*dl;
    }
    out_d_l[row*out_w + column] = dl;
  }
}

__global__ void update_layer(int w, int h, int batch_size, float lr, float* weights, float* biases, float* activations, float* d_l)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    float dw = 0.f;
    float db = 0.f;
    for(int i = 0; i < batch_size; i++)
    {
      float act = activations[i*h + row];
      float dl = d_l[i*w + column];
      dw += act*dl;
      db += dl;
    }
    weights[row*w + column] -= lr * dw / batch_size;
    biases[column] -= lr * db / batch_size;
  }
}

__global__ void relu(int w, int h, float* a, float* b)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < w && column < h)
  {
    float activation = a[row*w+column];
    b[row*w+column] =  activation > 0.f ? activation : 0.f;
  }
}

__global__ void relu_backwards(int w, int h, int ns, float* a, float* d_l, float* b)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < w && column < h)
  {
    float activation = a[row*w+column];
    b[row*w+column] = activation > 0.f ? d_l[row*w+column] : 0.f;
  }
}

__global__ void softmax(int w, int h, float* a, float* b)
{
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    float maxval = a[row*w];
    for (int i = 1; i<w; i++)
    {
      maxval = max(maxval, a[row*w + i]);
    }
    float divisor = 0.f;
    for (int i = 0; i<w; i++)
    {
      divisor += exp(a[row*w + i] - maxval);
    }
    b[row*w + col] = exp(a[row*w + col]-maxval)/(divisor);
    
  }
}

__global__ void cross_entropy(int w, int h, float* preds, float* real, float* output)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < h)
  {
    float loss = 0.f;
    for (int i = 0; i<w; i++)
    {
      loss -= real[idx*w + i] * log(max(1e-6, preds[idx*w + i]));
    }
    output[idx] = loss;
  }
}

__global__ void cross_entropy_backwards(int w, int h, float* preds, float* real, float* output)
{
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    output[row*w + col] = preds[row*w + col] - real[row*w + col];
  }
}

__global__ void init_rand(int w, int h, float* mat)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    hiprandState state;
    hiprand_init(44, row*w+column, 0, &state);
    mat[row*w + column] = hiprand_normal(&state)*sqrtf(2.f/h);
  }
}

void print_matrix(int w, int h, float* matrix, std::string title)
{
  float* m_h = new float[w*h];
  hipMemcpy(m_h, matrix, w*h*sizeof(float), hipMemcpyDeviceToHost);
  std::cout<<title<<std::endl;
  for(int i = 0; i<h; i++)
  {
    for(int j = 0; j<w; j++)
    {
      std::cout<<std::fixed<<std::setprecision(3)<<m_h[i*w+j]<<", ";
    }
    std::cout<<std::endl;
  }
  free(m_h);
}

void initLayer(float* weights, float* biases, int w, int h, int BLOCK_SIZE)
{
  dim3 dimGrid = dim3(ceil(w/(float)BLOCK_SIZE), ceil(h/(float)BLOCK_SIZE), 1);
  dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
  init_rand<<<dimGrid, dimBlock>>>(w, h, weights);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  dimGrid = dim3(ceil(h/(float)BLOCK_SIZE), 1, 1);
  dimBlock = dim3(BLOCK_SIZE, 1, 1);
  init_rand<<<dimGrid, dimBlock>>>(1, h, biases);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
}


void test_forward()
{
  int N = 1024;
  float* input = new float[N*N];
  float* weights = new float[N*N];
  float* biases = new float[N];
  float* output = new float[N*N];
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dist(-1.f, 1.f);
  for (int i = 0; i<N; i++)
  {
    weights[i*N + i] = 2;
    for (int j = 0; j<N; j++)
    {
      input[i*N + j] = dist(gen);
    }
    biases[i] = dist(gen);
  }
  float* weights_d;
  float* biases_d;
  float* input_d;
  float* output_d;

  int BLOCK_SIZE = 16;
  dim3 dimGrid = dim3(ceil(N/(float)BLOCK_SIZE), ceil(N/(float)BLOCK_SIZE), 1);
  dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

  gpuErrchk(hipMalloc((void**) &input_d, N*N*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &weights_d, N*N*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &output_d, N*N*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &biases_d, N*sizeof(float)));

  gpuErrchk(hipMemcpy(weights_d, weights, N*N*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(input_d, input, N*N*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(biases_d, biases, N*sizeof(float), hipMemcpyHostToDevice));

  forward<<<dimGrid, dimBlock>>>(N, N, N, input_d, weights_d, biases_d, output_d);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(output, output_d, N*N*sizeof(float), hipMemcpyDeviceToHost));
  for(int i = 0; i<N; i++)
  {
    for(int j = 0; j<N; j++)
    {
      float out = output[i*N+j];
      float expected = 2*input[i*N+j] + biases[j];
      ASSERT(out == expected, "INVALID at %d,%d, got %f, expected %f", i, j, out, expected);
    }
  }
  hipFree(output_d);
  hipFree(weights_d);
  hipFree(biases_d);
  hipFree(input_d);

  free(output);
  free(weights);
  free(biases);
  free(input);
}

void test_relu()
{
  int N = 2;
  float* input = new float[N*N];
  input[0] = 1.f;
  input[1] = -1.f;
  input[2] = 2.f;
  input[3] = 0.f;
  float* output = new float[N*N];
  float* input_d;
  float* output_d;

  int BLOCK_SIZE = 16;
  dim3 dimGrid = dim3(ceil(N/(float)BLOCK_SIZE), ceil(N/(float)BLOCK_SIZE), 1);
  dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

  gpuErrchk(hipMalloc((void**) &input_d, N*N*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &output_d, N*N*sizeof(float)));

  gpuErrchk(hipMemcpy(input_d, input, N*N*sizeof(float), hipMemcpyHostToDevice));

  relu<<<dimGrid, dimBlock>>>(N, N, input_d, output_d);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(output, output_d, N*N*sizeof(float), hipMemcpyDeviceToHost));

  ASSERT(output[0] == 1.f, "INVALID at %d, got %f, expected %f", 0, output[0], 1.f);
  ASSERT(output[1] == 0.f, "INVALID at %d, got %f, expected %f", 1, output[1], 0.f);
  ASSERT(output[2] == 2.f, "INVALID at %d, got %f, expected %f", 2, output[2], 2.f);
  ASSERT(output[3] == 0.f, "INVALID at %d, got %f, expected %f", 3, output[3], 0.f);
  hipFree(output_d);
  hipFree(input_d);

  free(output);
  free(input);
}

void test_softmax()
{
  int W = 3;
  int H = 4;
  float tolerance = 1e-6;
  float* input = new float[W*H];
  input[0] = 1.f;
  input[1] = 3.f;
  input[2] = 2.f;

  input[3] = -1.f;
  input[4] = 3.f;
  input[5] = 0.f;

  input[6] = 1.f;
  input[7] = 1.f;
  input[8] = 1.f;

  input[9] = 2.f;
  input[10] = 2.f;
  input[11] = 2.f;
  float* output = new float[W*H];
  float* expected = new float[W*H];

  expected[0] = 0.090031f;
  expected[1] = 0.665241f;
  expected[2] = 0.244728f;

  expected[3] = 0.017148f;
  expected[4] = 0.93624f;
  expected[5] = 0.046613f;

  expected[6] = 0.333333f;
  expected[7] = 0.333333f;
  expected[8] = 0.333333f;

  expected[9] = 0.333333f;
  expected[10] = 0.333333f;
  expected[11] = 0.333333f;
  float* input_d;
  float* output_d;

  int BLOCK_SIZE = 16;
  dim3 dimGrid = dim3(ceil(W/(float)BLOCK_SIZE), ceil(H/(float)BLOCK_SIZE), 1);
  dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

  gpuErrchk(hipMalloc((void**) &input_d, W*H*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &output_d, W*H*sizeof(float)));

  gpuErrchk(hipMemcpy(input_d, input, W*H*sizeof(float), hipMemcpyHostToDevice));

  softmax<<<dimGrid, dimBlock>>>(W, H, input_d, output_d);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(output, output_d, W*H*sizeof(float), hipMemcpyDeviceToHost));
  for(int i = 0; i<W*H; i++)
  {
    ASSERT(abs(output[i] - expected[i]) < tolerance, "failed at %d, expected %f, got %f", i, output[i], expected[i]);
  }

  hipFree(output_d);
  hipFree(input_d);

  free(output);
  free(input);
  free(expected);
}

void test_crossentropy()
{
  int W = 3;
  int H = 4;
  float tolerance = 1e-6;
  float* preds = new float[W*H];
  preds[0] = 0.05f;
  preds[1] = 0.9f;
  preds[2] = 0.05f;

  preds[3] = 0.3f;
  preds[4] = 0.3f;
  preds[5] = 0.4f;

  preds[6] = 0.99f;
  preds[7] = 0.05f;
  preds[8] = 0.05f;

  preds[9] = 0.99f;
  preds[10] = 0.05f;
  preds[11] = 0.05f;

  float* output = new float[H];
  float* expected = new float[H];
  expected[0] = 2.995732f;
  expected[1] = 1.203973f;
  expected[2] = 2.995732f;
  expected[3] = 0.010050f;
  float* real = new float[W*H];

  real[0] = 1;
  real[1] = 0;
  real[2] = 0;

  real[3] = 0;
  real[4] = 1;
  real[5] = 0;

  real[6] = 0;
  real[7] = 0;
  real[8] = 1;

  real[9] = 1;
  real[10] = 0;
  real[11] = 0;
  float* preds_d;
  float* real_d;
  float* output_d;

  int BLOCK_SIZE = 16;
  dim3 dimGrid = dim3(ceil(H/(float)BLOCK_SIZE), 1, 1);
  dim3 dimBlock = dim3(BLOCK_SIZE, 1, 1);

  gpuErrchk(hipMalloc((void**) &preds_d, W*H*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &real_d, W*H*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &output_d, H*sizeof(float)));

  gpuErrchk(hipMemcpy(preds_d, preds, W*H*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(real_d, real, W*H*sizeof(float), hipMemcpyHostToDevice));

  cross_entropy<<<dimGrid, dimBlock>>>(W, H, preds_d, real_d, output_d);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(output, output_d, H*sizeof(float), hipMemcpyDeviceToHost));
  for(int i = 0; i<H; i++)
  {
    ASSERT(abs(output[i] - expected[i]) < tolerance, "failed at %d, expected %f, got %f", i, expected[i], output[i]);
  }

  hipFree(output_d);
  hipFree(preds_d);
  hipFree(real_d);

  free(output);
  free(preds);
  free(real);
  free(expected);
}

void test()
{
  std::cout<<"running test forward"<<std::endl;
  test_forward();
  std::cout<<"running test relu"<<std::endl;
  test_relu();
  std::cout<<"running test softmax"<<std::endl;
  test_softmax();
  std::cout<<"running test crossentropy"<<std::endl;
  test_crossentropy();

}

void read_mnist(const std::string filename, int length, float* x, float* y)
{
  int input_size = 784;
  int labels = 10;

  std::fstream fin;
  fin.open("./mnist_train.csv");
  std::string row;
  constexpr char delim = ',';
  for(int i = 0; i<length; i++)
  {
    fin >> row;
    int pos = row.find(delim);
    int label = std::stoi(row.substr(0, pos+1));
    for(int j = 0; j<labels; j++)
    {
      y[labels*i + j] = (j==label);
    }
    row.erase(0, pos+1);
    for(int j = 0; j<input_size; j++)
    {
      pos = row.find(delim);
      if (pos == std::string::npos)
      {
        pos = row.length() - 1;
      }
      x[i*input_size+j] = std::stof(row.substr(0, pos+1)) / 255; //normalize value
      row.erase(0, pos+1);
    }
    ASSERT(row.length() == 0, "didn't parse all values in row, %d", i);
  }
}

int main(int argc, char** argv)
{
  if (argc > 1 && std::string(argv[1]) == "--test")
  {
    test();
    return 0;
  }

  int test_length = 10000;
  int train_length = 60000;

  float* input;
  float* labels;
  int input_size = 784;
  int labels_size = 10;

  float* mnist_train_x = new float[input_size * train_length];
  float* mnist_train_y = new float[labels_size * train_length];
  read_mnist("./mnist_train.csv", train_length, mnist_train_x, mnist_train_y);

  float* mnist_test_x = new float[input_size * test_length];
  float* mnist_test_y = new float[labels_size * test_length];
  read_mnist("./mnist_test.csv", test_length, mnist_test_x, mnist_test_y);

  int size1 = 100;
  float* weights1;
  float* biases1;
  float* d_l1;

  int size2 = 30;
  float* weights2;
  float* biases2;
  float* d_l2;

  int size3 = 10;
  float* weights3;
  float* biases3;
  float* d_l3;

  int BLOCK_SIZE = 16;
  int BATCH_SIZE = 16;
  int EPOCHS = 3000;
  float LR = 3.f;
  dim3 dimGrid;
  dim3 dimBlock;


  gpuErrchk(hipMalloc((void**) &input, input_size*BATCH_SIZE*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &labels, labels_size*BATCH_SIZE*sizeof(float)));

  gpuErrchk(hipMalloc((void**) &weights1, size1*input_size*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &biases1, size1*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &d_l1, input_size*BATCH_SIZE*sizeof(float)));
  initLayer(weights1, biases1, size1, input_size, BLOCK_SIZE);

  gpuErrchk(hipMalloc((void**) &weights2, size2*size1*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &biases2, size2*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &d_l2, size1*BATCH_SIZE*sizeof(float)));
  initLayer(weights2, biases2, size2, size1, BLOCK_SIZE);


  gpuErrchk(hipMalloc((void**) &weights3, size3*size2*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &biases3, size3*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &d_l3, size2*BATCH_SIZE*sizeof(float)));
  initLayer(weights3, biases3, size3, size2, BLOCK_SIZE);

  float *x1;
  float *a1;
  gpuErrchk(hipMalloc((void**) &x1, size1*BATCH_SIZE*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &a1, size1*BATCH_SIZE*sizeof(float)));

  float *x2;
  float *a2;
  gpuErrchk(hipMalloc((void**) &x2, size2*BATCH_SIZE*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &a2, size2*BATCH_SIZE*sizeof(float)));

  float *x3;
  float *a3;
  gpuErrchk(hipMalloc((void**) &x3, size3*BATCH_SIZE*sizeof(float)));
  gpuErrchk(hipMalloc((void**) &a3, size3*BATCH_SIZE*sizeof(float)));

  for(int epoch = 0; epoch<EPOCHS; epoch++)
  {
    float cum_loss = 0.f;
    int correct = 0;
    int total = 0;
    for(int batch = 0; batch<train_length/BATCH_SIZE; batch++)
    {
      total += BATCH_SIZE;
      gpuErrchk(hipMemcpy(input, &mnist_train_x[batch*BATCH_SIZE*input_size], BATCH_SIZE*input_size*sizeof(float), hipMemcpyHostToDevice)); 
      gpuErrchk(hipMemcpy(labels, &mnist_train_y[batch*BATCH_SIZE*labels_size], BATCH_SIZE*labels_size*sizeof(float), hipMemcpyHostToDevice)); 

      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, input_size, size1, input, weights1, biases1, x1);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());

      relu<<<dimGrid, dimBlock>>>(size1, BATCH_SIZE, x1, a1);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());

      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size1, size2, a1, weights2, biases2, x2);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());

      relu<<<dimGrid, dimBlock>>>(size2, BATCH_SIZE, x2, a2);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      forward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size2, size3, a2, weights3, biases3, x3);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());

      softmax<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, x3, a3);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      
      float* loss;
      gpuErrchk(hipMalloc((void**) &loss, BATCH_SIZE*sizeof(float)));

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), 1, 1);
      dimBlock = dim3(BLOCK_SIZE, 1, 1);
      cross_entropy<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, a3, labels, loss);

      float* out_h = new float[BATCH_SIZE*size3];
      gpuErrchk(hipMemcpy(out_h, a3, BATCH_SIZE*size3*sizeof(float), hipMemcpyDeviceToHost));

      float* loss_h = new float[BATCH_SIZE];
      gpuErrchk(hipMemcpy(loss_h, loss, BATCH_SIZE*sizeof(float), hipMemcpyDeviceToHost));
      
      for (int i = 0; i < BATCH_SIZE; i++)
      {
        float max_1 = 0.f;
        float max_2 = 0.f;
        int i1 = 0;
        int i2 = 0;
        for (int j = 0; j<labels_size; j++)
        {
          if (out_h[i*labels_size + j] > max_1)
          {
            max_1 = out_h[i*labels_size + j];
            i1 = j;
          }
          
          if (mnist_train_y[batch*BATCH_SIZE*labels_size + i*labels_size + j] > max_2)
          {
            max_2 = mnist_train_y[batch*BATCH_SIZE*labels_size + i*labels_size + j];
            i2 = j;
          }
        }
        correct += (i1 == i2);
        cum_loss += loss_h[i];
      }

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      float* d_L;
      gpuErrchk(hipMalloc((void**) &d_L, size3*BATCH_SIZE*sizeof(float)));

      cross_entropy_backwards<<<dimGrid, dimBlock>>>(size3, BATCH_SIZE, a3, labels, d_L);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());

      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      backward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size3, size2, LR, weights3, biases3, d_L, d_l3);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());

      relu_backwards<<<dimGrid, dimBlock>>>(size2, BATCH_SIZE, size3, a2, d_l3, d_l3);

      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(BATCH_SIZE/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

      backward<<<dimGrid, dimBlock>>>(BATCH_SIZE, size2, size1, LR, weights2, biases2, d_l3, d_l2);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      relu_backwards<<<dimGrid, dimBlock>>>(size1, BATCH_SIZE, size2, a1, d_l2, d_l2);

      dimGrid = dim3(ceil(size3/(float)BLOCK_SIZE), ceil(size2/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
      update_layer<<<dimGrid, dimBlock>>>(size3, size2, BATCH_SIZE, LR, weights3, biases3, a2, d_L);
      dimGrid = dim3(ceil(size2/(float)BLOCK_SIZE), ceil(size1/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
      update_layer<<<dimGrid, dimBlock>>>(size2, size1, BATCH_SIZE, LR, weights2, biases2, a1, d_l3);
      dimGrid = dim3(ceil(size1/(float)BLOCK_SIZE), ceil(input_size/(float)BLOCK_SIZE), 1);
      dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
      update_layer<<<dimGrid, dimBlock>>>(size1, input_size, BATCH_SIZE, LR, weights1, biases1, input, d_l2);

    }
    std::cout<<"epoch "<<epoch<<" cum loss "<<cum_loss<<" accuracy "<< (float)correct/total<<std::endl;
  }
}
