
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <ostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void matvec(int n, float* a, float* b, float* c)
{
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  if (col < n)
  {
    for(int i = 0; i < n; i++)
    {
      c[col] += a[i*n + col] * b[col];
    }
  }
}


int main()
{
  int N = 1024;
  int BLOCK_SIZE=32;
  float* a = new float[N*N];
  float* b = new float[N];
  float* c = new float[N];
  for (int i = 0; i<N; i++)
  {
    b[i] = i;
    a[i*N + i] = 2;
  }
  float* a_d;
  float* b_d;
  float* c_d;

  hipMalloc((void**) &a_d, N*N*sizeof(float));
  hipMalloc((void**) &b_d, N*sizeof(float));
  hipMalloc((void**) &c_d, N*sizeof(float));

  hipMemcpy(a_d, a, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, N*sizeof(float), hipMemcpyHostToDevice);

  dim3 dimGrid(ceil(N/(float)BLOCK_SIZE),1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  matvec<<<dimGrid, dimBlock>>>(N, a_d, b_d, c_d);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpy(c, c_d, N*sizeof(float), hipMemcpyDeviceToHost);

  
  for (int i = 0; i<N; i++)
  {
    assert(c[i] == b[i]*2);
  }
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
  return 0;
}
