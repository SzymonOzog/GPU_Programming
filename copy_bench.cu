
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <cassert>
#include <hipblas.h>

#define BLOCK_SIZE 128 
#define BENCH_STEPS 1000
 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))

using datatype = half;
using datatype_vec = half2;

// using datatype = float;
// using datatype_vec = float4;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void clear_l2() {
    // Get actual L2 size via CUDA on first call of this function
    static int l2_clear_size = 0;
    static unsigned char* gpu_scratch_l2_clear = NULL;
    if (!gpu_scratch_l2_clear) {
        hipDeviceGetAttribute(&l2_clear_size, hipDeviceAttributeL2CacheSize, 0);
        l2_clear_size *= 2; // just to be extra safe (cache is not necessarily strict LRU)
        gpuErrchk(hipMalloc(&gpu_scratch_l2_clear, l2_clear_size));
    }
    // Clear L2 cache (this is run on every call unlike the above code)
    gpuErrchk(hipMemset(gpu_scratch_l2_clear, 0, l2_clear_size));
}

__global__ void copy(int n , datatype* in, datatype* out)
{
  unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    out[i] = in[i];
  }
}

__global__ void copyf4(int n , datatype_vec* in, datatype_vec* out)
{
  unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    out[i] = in[i];
  }
}

__global__ void copy_loop(int n , datatype* in, datatype* out, int max_size)
{
  unsigned long i = blockIdx.x * blockDim.x;
  for (int idx = i * max_size; idx < (i+blockDim.x)*max_size; idx+=blockDim.x)
  {
      if (idx<n)
      {
        out[idx+threadIdx.x] = in[idx+threadIdx.x];
      }
  }
}

__global__ void copy_loop_datatype4(int n , datatype_vec* in, datatype_vec* out, int max_size)
{
  unsigned long i = blockIdx.x * blockDim.x;
  for (int idx = i * max_size; idx < (i+blockDim.x)*max_size; idx+=blockDim.x)
  {
      if (idx<n)
      {
        out[idx+threadIdx.x] = in[idx+threadIdx.x];
      }
  }
}

int main()
{
  datatype* in_d;
  datatype* out_d;
  datatype* out2_d;

  long N = std::pow<long, long>(2, 25);

    //one warmup run
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    dim3 dimGrid(ceil(N/(float)BLOCK_SIZE), 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    hipMalloc((void**) &in_d, N*sizeof(datatype));
    datatype* cp = new datatype[N];
    for (int i = 0; i < N; i++)
    {
        cp[i] = (float)N;
    }
    hipMemcpy(in_d, cp, N*sizeof(datatype), hipMemcpyHostToDevice);

    hipMalloc((void**) &out_d, N*sizeof(datatype));
    hipMemset(out_d, 0, N*sizeof(datatype));

    hipMalloc((void**) &out2_d, N*sizeof(datatype));
    hipMemset(out2_d, 0, N*sizeof(datatype));
    float time = 0.f;
    double run_time = 0.0;
    for (int i = -1; i<BENCH_STEPS; i++)
    {
      clear_l2();
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(start));
      copy<<<dimGrid, dimBlock>>>(N, in_d, out_d);
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time, start, stop));
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      if (i != -1) // one warmup run
      {
        run_time += time / BENCH_STEPS;
      }
    }

    std::cout<<"regular time "<<run_time<<std::endl;

    dimGrid = dim3(ceil(N/(float)(BLOCK_SIZE*4)), 1, 1);
    dimBlock = dim3(BLOCK_SIZE, 1, 1);
    time = 0.f;
    run_time = 0.0;
    for (int i = -1; i<BENCH_STEPS; i++)
    {
      clear_l2();
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(start));
      copyf4<<<dimGrid, dimBlock>>>(N/4, reinterpret_cast<datatype_vec*>(in_d), reinterpret_cast<datatype_vec*>(out_d));
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time, start, stop));
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      if (i != -1) // one warmup run
      {
        run_time += time / BENCH_STEPS;
      }
    }

    std::cout<<"vectorized time "<<run_time<<std::endl;

    int loop_size = 1024;
    dimGrid = dim3(ceil(N/(float)(BLOCK_SIZE*loop_size)), 1, 1);
    dimBlock = dim3(BLOCK_SIZE, 1, 1);
    time = 0.f;
    run_time = 0.0;
    for (int i = -1; i<BENCH_STEPS; i++)
    {
      clear_l2();
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(start));
      copy_loop<<<dimGrid, dimBlock>>>(N, in_d, out2_d, loop_size);
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time, start, stop));
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      if (i != -1) // one warmup run
      {
        run_time += time / BENCH_STEPS;
      }
    }

    datatype* out_h = new datatype[N];
    datatype* out2_h = new datatype[N];
    hipMemcpy(out_h, out_d, N*sizeof(datatype), hipMemcpyDeviceToHost);
    hipMemcpy(out2_h, out2_d, N*sizeof(datatype), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
      ASSERT(out_h[i] == out2_h[i], "failed at copy loop %d, %f, %f\n", i, (float)out_h[i], (float)out2_h[i]);
    }

    loop_size = loop_size/4;
    std::cout<<"loop time "<<run_time<<std::endl;
    dimGrid = dim3(ceil(N/(float)(BLOCK_SIZE*4*loop_size)), 1, 1);
    dimBlock = dim3(BLOCK_SIZE, 1, 1);
    time = 0.f;
    run_time = 0.0;
    for (int i = -1; i<BENCH_STEPS; i++)
    {
      clear_l2();
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(start));
      copy_loop_datatype4<<<dimGrid, dimBlock>>>(N/4, reinterpret_cast<datatype_vec*>(in_d), reinterpret_cast<datatype_vec*>(out2_d), loop_size);
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time, start, stop));
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      if (i != -1) // one warmup run
      {
        run_time += time / BENCH_STEPS;
      }
    }
    std::cout<<"loop time vectorized "<<run_time<<std::endl;

    hipMemcpy(out_h, out_d, N*sizeof(datatype), hipMemcpyDeviceToHost);
    hipMemcpy(out2_h, out2_d, N*sizeof(datatype), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
      ASSERT(out_h[i] == out2_h[i], "failed at  copy loopa datatype 4 %d, %f, %f\n", i, (float)out_h[i], (float)out2_h[i]);
    }

  hipFree(in_d);
  hipFree(out_d);
  return 0;
}

